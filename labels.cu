#include "labels.h"

namespace kmeans {
namespace detail {

hipblasHandle_t cublas_handle;

void labels_init() {
    hipblasStatus_t stat;
    stat = hipblasCreate(&detail::cublas_handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "CUBLAS initialization failed" << std::endl;
        exit(1);
    }
}

void gemm(hipblasOperation_t transa,
          hipblasOperation_t transb,
          int m, int n, int k,
          const float *alpha,
          const float *A, int lda,
          const float *B, int ldb,
          const float *beta,
          float *C, int ldc) {
    hipblasStatus_t status = hipblasSgemm(cublas_handle, transa, transb,
                                        m, n, k, alpha,
                                        A, lda,
                                        B, ldb,
                                        beta,
                                        C, ldc);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "Invalid Sgemm" << std::endl;
        exit(1);
    }
}

void gemm(hipblasOperation_t transa,
                    hipblasOperation_t transb,
                    int m, int n, int k,
                    const double *alpha,
                    const double *A, int lda,
                    const double *B, int ldb,
                    const double *beta,
                    double *C, int ldc) {
    hipblasStatus_t status = hipblasDgemm(cublas_handle, transa, transb,
                                        m, n, k, alpha,
                                        A, lda,
                                        B, ldb,
                                        beta, 
                                        C, ldc);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "Invalid Sgemm" << std::endl;
        exit(1);
    }
}

}
}
