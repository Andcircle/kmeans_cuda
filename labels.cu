#include "hip/hip_runtime.h"
#include "labels.h"
#include <hipblas.h>
#include <cfloat>

namespace kmeans {
namespace detail {

hipblasHandle_t cublas_handle;

void labels_init() {
    hipblasStatus_t stat;
    stat = hipblasCreate(&detail::cublas_handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "CUBLAS initialization failed" << std::endl;
        exit(1);
    }
}

//n: number of points
//d: dimensionality of points
//data: points, laid out in row-major order (n rows, d cols)
//dots: result vector (n rows)
// NOTE:
//Memory accesses in this function are uncoalesced!!
//This is because data is in row major order
//However, in k-means, it's called outside the optimization loop
//on the large data array, and inside the optimization loop it's
//called only on a small array, so it doesn't really matter.
//If this becomes a performance limiter, transpose the data somewhere
__global__ void self_dots(int n, int d, double* data, double* dots) {
	double accumulator = 0;
    int global_id = blockDim.x * blockIdx.x + threadIdx.x;

    if (global_id < n) {
        for (int i = 0; i < d; i++) {
            double value = data[i + global_id * d];
            accumulator += value * value;
        }
        dots[global_id] = accumulator;
    }    
}


void make_self_dots(int n, int d, thrust::device_vector<double>& data, thrust::device_vector<double>& dots) {
    self_dots<<<(n-1)/256+1, 256>>>(n, d, thrust::raw_pointer_cast(data.data()),
                                    thrust::raw_pointer_cast(dots.data()));
}

__global__ void all_dots(int n, int k, double* data_dots, double* centroid_dots, double* dots) {
	__shared__ double local_data_dots[32];
	__shared__ double local_centroid_dots[32];

    int data_index = threadIdx.x + blockIdx.x * blockDim.x;
    if ((data_index < n) && (threadIdx.y == 0)) {
        local_data_dots[threadIdx.x] = data_dots[data_index];
    }

    

    
    int centroid_index = threadIdx.x + blockIdx.y * blockDim.y;
    if ((centroid_index < k) && (threadIdx.y == 1)) {
        local_centroid_dots[threadIdx.x] = centroid_dots[centroid_index];
    }
       
   	__syncthreads();

	centroid_index = threadIdx.y + blockIdx.y * blockDim.y;
    if ((data_index < n) && (centroid_index < k)) {
        dots[data_index + centroid_index * n] = local_data_dots[threadIdx.x] +
            local_centroid_dots[threadIdx.y];
    }
}

void make_all_dots(int n, int k, thrust::device_vector<double>& data_dots,
                   thrust::device_vector<double>& centroid_dots,
                   thrust::device_vector<double>& dots) {
    all_dots<<<
        dim3((n-1)/32+1,
             (k-1)/32+1),
        dim3(32, 32)>>>(n, k, thrust::raw_pointer_cast(data_dots.data()),
                        thrust::raw_pointer_cast(centroid_dots.data()),
                        thrust::raw_pointer_cast(dots.data()));
};

void calculate_distances(int n, int d, int k,
                         thrust::device_vector<double>& data,
                         thrust::device_vector<double>& centroids,
                         thrust::device_vector<double>& data_dots,
                         thrust::device_vector<double>& centroid_dots,
                         thrust::device_vector<double>& pairwise_distances) {
    detail::make_self_dots(k, d, centroids, centroid_dots);
    detail::make_all_dots(n, k, data_dots, centroid_dots, pairwise_distances);
    //||x-y||^2 = ||x||^2 + ||y||^2 - 2 x . y
    //pairwise_distances has ||x||^2 + ||y||^2, so beta = 1
    //The dgemm calculates x.y for all x and y, so alpha = -2.0
    double alpha = -2.0;
    double beta = 1.0;
    //If the data were in standard column major order, we'd do a
    //centroids * data ^ T
    //But the data is in row major order, so we have to permute
    //the arguments a little
    hipblasStatus_t stat =
        hipblasDgemm(detail::cublas_handle,
                    HIPBLAS_OP_T, HIPBLAS_OP_N,
                    n, k, d, &alpha,
                    thrust::raw_pointer_cast(data.data()),
                    d,//Has to be n or d
                    thrust::raw_pointer_cast(centroids.data()),
                    d,//Has to be k or d
                    &beta,
                    thrust::raw_pointer_cast(pairwise_distances.data()),
                    n); //Has to be n or k
    
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "Invalid Dgemm" << std::endl;
        exit(1);
    }

}
                         
__global__ void make_new_labels(int n, int k, double* pairwise_distances,
                                int* labels, int* changes,
                                double* distances) {
    double min_distance = DBL_MAX;
    double min_idx = -1;
    int global_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (global_id < n) {
        int old_label = labels[global_id];
        for(int c = 0; c < k; c++) {
            double distance = pairwise_distances[c * n + global_id];
            if (distance < min_distance) {
                min_distance = distance;
                min_idx = c;
            }
        }
        labels[global_id] = min_idx;
        distances[global_id] = min_distance;
        if (old_label != min_idx) {
            atomicAdd(changes, 1);
        }
    }
}


int relabel(int n, int k,
            thrust::device_vector<double>& pairwise_distances,
            thrust::device_vector<int>& labels,
            thrust::device_vector<double>& distances) {
    thrust::device_vector<int> changes(1);
    changes[0] = 0;
    make_new_labels<<<(n-1)/256+1,256>>>(
        n, k,
        thrust::raw_pointer_cast(pairwise_distances.data()),
        thrust::raw_pointer_cast(labels.data()),
        thrust::raw_pointer_cast(changes.data()),
        thrust::raw_pointer_cast(distances.data()));
    return changes[0];
}

}
}
