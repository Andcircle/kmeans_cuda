#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <vector>
#include <math.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "helpers.h"

#include "../matplotlibcpp.h"
#include <vector>

namespace plt = matplotlibcpp;

__device__ float
squared_l2_distance(int d, float* point, float* centroid) {
  float dist = 0;
  for (int i=0; i<d; ++i)
    dist += pow((point[i] - centroid[i]), 2);
  return dist;
}

// In the assignment step, each point (thread) computes its distance to each
// cluster centroid and adds its x and y values to the sum of its closest
// centroid, as well as incrementing that centroid's count of assigned points.
__global__ void assign_clusters(int data_size,
                                int d,
                                int k,
                                const thrust::device_ptr<float> data,
                                const thrust::device_ptr<float> means,
                                thrust::device_ptr<float> new_sums,
                                thrust::device_ptr<int> counts) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= data_size) return;

  // Make global loads once.
  float* point = thrust::raw_pointer_cast(data + index * d);

  float best_distance = FLT_MAX;
  int best_cluster = 0;
  for (int cluster = 0; cluster < k; ++cluster) {
    float* centroid = thrust::raw_pointer_cast(means + cluster * d);
    const float distance =
        squared_l2_distance(d, point, centroid);
    if (distance < best_distance) {
      best_distance = distance;
      best_cluster = cluster;
    }
  }

  for (int i=0; i<d; ++i)
    atomicAdd(thrust::raw_pointer_cast(new_sums + best_cluster * d + i), point[i]);
  atomicAdd(thrust::raw_pointer_cast(counts + best_cluster), 1);
}

// Each thread is one cluster, which just recomputes its coordinates as the mean
// of all points assigned to it.
__global__ void compute_new_means(int d,
                                  thrust::device_ptr<float> means,
                                  const thrust::device_ptr<float> new_sums,
                                  const thrust::device_ptr<int> counts) {
  const int cluster = threadIdx.x;
  const int count = max(1, counts[cluster]);
  
  for (int i=0; i<d; ++i)
    means[cluster * d + i] = new_sums[cluster * d + i] / count;
}

int main(int argc, const char* argv[]) {
  // if (argc != 5) {
  //   std::cerr << "usage: executable <number of data points> <number of cluster> <data dimension> <iteration>"
  //             << std::endl;
  //   std::exit(EXIT_FAILURE);
  // }

  // const int n = std::atoi(argv[1]);
  // const int k = std::atoi(argv[2]);
  // const int d = std::atoi(argv[3]);
  // const int iterations = std::atoi(argv[4]);

  int iterations = 50;
    int n = 1e6;
    int d = 64;
    int k = 128;

  thrust::device_vector<float> d_data(n * d);

  random_data(d_data, n, d);
  thrust::device_vector<float> d_mean(d_data.begin(), d_data.begin() + k * d);

  // float dataset[] = {
  //   0.5, 0.5,
  //   1.5, 0.5,
  //   1.5, 1.5,
  //   0.5, 1.5,
  //   1.1, 1.2,
  //   0.5, 15.5,
  //   1.5, 15.5,
  //   1.5, 16.5,
  //   0.5, 16.5,
  //   1.2, 16.1,
  //   15.5, 15.5,
  //   16.5, 15.5,
  //   16.5, 16.5,
  //   15.5, 16.5,
  //   15.6, 16.2,
  //   15.5, 0.5,
  //   16.5, 0.5,
  //   16.5, 1.5,
  //   15.5, 1.5,
  //   15.7, 1.6};
  // float centers[] = {
  //   0.5, 0.5,
  //   1.5, 0.5,
  //   1.5, 1.5,
  //   0.5, 1.5};
   
  //   int iterations = 3;
  //   int n = 20;
  //   int d = 2;
  //   int k = 4;
  
  // thrust::device_vector<float> d_data(dataset, dataset+n*d);
  // thrust::device_vector<float> d_mean(centers, centers+k*d);

  // std::vector<float> x;
  // std::vector<float> y;

  // for (size_t pos = 0; pos < n; ++pos) {
  //   x.push_back(d_data[pos*2]);
  //   y.push_back(d_data[pos*2+1]);
  // }

  // plt::plot(x, y,  {{"color", "blue"}, {"marker", "."}, {"linestyle", ""}});

  thrust::device_vector<float> d_sums(k * d);
  thrust::device_vector<int> d_counts(k, 0);

  const int threads = 1024;
  const int blocks = (n + threads - 1) / threads;

  const auto start = std::chrono::high_resolution_clock::now();
  for (size_t iteration = 0; iteration < iterations; ++iteration) {
    thrust::fill(d_sums.begin(), d_sums.end(), 0);
    thrust::fill(d_counts.begin(), d_counts.end(), 0);

    assign_clusters<<<blocks, threads>>>(n, d, k,
                                         d_data.data(),
                                         d_mean.data(),
                                         d_sums.data(),
                                         d_counts.data());
    hipDeviceSynchronize();

    compute_new_means<<<1, k>>>(d,
                                d_mean.data(),
                                d_sums.data(),
                                d_counts.data());
    hipDeviceSynchronize();
  }
  const auto end = std::chrono::high_resolution_clock::now();
  const auto duration =
      std::chrono::duration_cast<std::chrono::duration<float>>(end - start);
  std::cerr << "Took: " << duration.count() << "s" << std::endl;

  for (size_t cluster = 0; cluster < k; ++cluster) {
    for (size_t i = 0; i < d; ++i){
      size_t idx = cluster * d + i;
      std::cout << d_mean[idx] << " ";
    }
    std::cout << std::endl;
  }

  // x.clear();
  // y.clear();

  // for (size_t c = 0; c < k; ++c) {
  //   x.push_back(d_mean[c*2]);
  //   y.push_back(d_mean[c*2+1]);
  // }

  // plt::plot(x, y,  {{"color", "red"}, {"marker", "o"}, {"linestyle", ""}});
  // plt::show();
}
